#include "hip/hip_runtime.h"
#include "to_bench.cuh"

#include "cuda_tools/cuda_error_checking.cuh"

#include <raft/core/device_span.hpp>

#include <rmm/device_uvector.hpp>
#include <rmm/device_scalar.hpp>


template <typename T>
__global__
void kernel_reduce_baseline(raft::device_span<const T> buffer, raft::device_span<T> total)
{
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < buffer.size())
        atomicAdd(total.data(), buffer[id]);
}

void baseline_reduce(rmm::device_uvector<int>& buffer,
                     rmm::device_scalar<int>& total)
{
    constexpr int blocksize = 64;
    const int gridsize = (buffer.size() + blocksize - 1) / blocksize;

	kernel_reduce_baseline<int><<<gridsize, blocksize, 0, buffer.stream()>>>(
        raft::device_span<int>(buffer.data(), buffer.size()),
        raft::device_span<int>(total.data(), 1));

    CUDA_CHECK_ERROR(hipStreamSynchronize(buffer.stream()));
}

template <typename T>
__global__
void kernel_your_reduce(raft::device_span<const T> buffer, raft::device_span<T> total)
{
    // TODO
    // ...
}

void your_reduce(rmm::device_uvector<int>& buffer,
                 rmm::device_scalar<int>& total)
{
    // TODO
    // ...

	kernel_your_reduce<int><<<1, 1, 0, buffer.stream()>>>(
        raft::device_span<const int>(buffer.data(), buffer.size()),
        raft::device_span<int>(total.data(), 1));

    CUDA_CHECK_ERROR(hipStreamSynchronize(buffer.stream()));
}